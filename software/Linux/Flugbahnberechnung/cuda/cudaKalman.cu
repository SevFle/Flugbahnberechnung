#include "hip/hip_runtime.h"
#include "cudaKalman.cuh"
#include <stdio.h>
#include <iostream>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

#define ALERT(cudaStat, message)   if (cudaStat != hipSuccess){ printf (message); return EXIT_FAILURE;}




using namespace cudaKalman;

C_cudaKalman::C_cudaKalman()
  {
  init();
  initMatrix(9,3,0);
  }
C_cudaKalman::~C_cudaKalman()
  {

  }
void C_cudaKalman::init               ()
  {
  std::cout << "hello from inti" << std::endl;
  hipblasCreate(&this->handle);
  hipblasSetStream(this->handle, this->streamkalman);

  }
void C_cudaKalman::deinit             ()
  {
  hipblasDestroy(this->handle);

  }

__host__ bool C_cudaKalman::initMatrix         (int dynamParams, int measureParams, int controlParams)
  {
  this->dynamParams = dynamParams;
  this->measureParams = measureParams;
  this->controlParams = controlParams;
  //Erzeuge neue Arrays
  std::cout << "Creating Host CUDA Matrices" << std::endl;
  //this->statePre= new float[dynamParams];


//  a = (float *)malloc (M * N * sizeof (*a));
//  if (!a) {
//      printf ("host memory allocation failed");
//      return EXIT_FAILURE;
//  }
//  for (j = 0; j < N; j++) {
//      for (i = 0; i < M; i++) {
//          a[IDX2C(i,j,M)] = (float)(i * M + j + 1);
//      }
//  }



  this->statePre = (float*)malloc(1*dynamParams*sizeof(this->statePre));
  this->statePost = (float*)malloc(1*dynamParams*sizeof(this->statePre));

  for (int j = 1; j <= dynamParams; j++)
    {
      for (int i = 1; i <= 1; i++)
        {
          this->statePre[IDX2C(i,j,1)] = (float)(i * 1 + j + 1);
          this->statePre[IDX2C(i,j,1)] = (float)(i * 1 + j + 1);
      }
    }
  for (int j = 1; j <= dynamParams; j++)
    {
      for (int i = 1; i <= 1; i++)
        {
          this->statePre[IDX2C(i,j,1)] = 0.0;
          this->statePre[IDX2C(i,j,1)] = 0.0;
      }
    }

  this->transitionMatrix = new float[dynamParams*dynamParams];
  this->transitionMatrix_temp = new float[dynamParams*dynamParams];

  this->transitionMatrix = (float*)malloc(dynamParams*dynamParams*sizeof(this->transitionMatrix));
  this->transitionMatrix_temp = (float*)malloc(dynamParams*dynamParams*sizeof(this->transitionMatrix_temp));


  this->processNoiseCov = new float[dynamParams*dynamParams];
  this->temp1 = new float[dynamParams*dynamParams];

  this->measurementMatrix = new float[measureParams*dynamParams];
  this->temp2 = new float[measureParams*dynamParams];
  this->temp2_temp = new float[measureParams*dynamParams];
  this->temp4 = new float[measureParams*dynamParams];
  for(int i=0;i<measureParams*dynamParams;i++)
    {
    this->measurementMatrix[i]=0.0;
    this->temp2[i]=0.0;
    this->temp4[i]=0.0;
    }

  this->measurementNoiseCov = new float[measureParams*measureParams];
  this->temp3 = new float[measureParams*measureParams];
  for(int i=0;i<measureParams*measureParams;i++)
    {
    this->measurementNoiseCov[i]=0.0;
    this->temp3[i]=0.0;
    }

  this->measurement = new float[measureParams];
  this->temp5 = new float[measureParams];
  for(int i=0;i<measureParams;i++)
    {
    this->measurement[i]=0.0;
    this->temp5[i]=0.0;
    }


  this->errorCovPre = new float[dynamParams*dynamParams];
  this->errorCovPost = new float[dynamParams*dynamParams];
  for(int i=0;i<dynamParams*dynamParams;i++)
    {
    this->errorCovPre[i]=0.0;
    this->errorCovPost[i]=0.0;
    this->transitionMatrix[i]=0.0;
    this->processNoiseCov[i]=0.0;
    }


  this->gain = new float[dynamParams*measureParams];
  if( controlParams > 0 )
    {
    this->controlMatrix = new float[dynamParams*controlParams];
    this->controlVector = new float[controlParams];
    }

  if (!statePre | !statePost |!transitionMatrix | !processNoiseCov |!measurementMatrix | !measurementNoiseCov |!measurement | !errorCovPre |!errorCovPost)
    {
    return false;
    }
  std::cout << "Creation of Host CUDA Matrices successful" << std::endl;


  //Schreibe die Initialwerte in die jeweiligen Matrizen
  std::cout << "Creating Device CUDA Matrices" << std::endl;

  cudaStat = hipMalloc ((void**)&statePre_devPtr, dynamParams*sizeof(*statePre));
  ALERT(cudaStat, "statePre device memory allocation failed");

  cudaStat = hipMalloc ((void**)&statePost_devPtr, dynamParams*sizeof(*statePost));
  ALERT(cudaStat, "statePost device memory allocation failed");

  cudaStat = hipMalloc ((void**)&transitionMatrix_devPtr, dynamParams*dynamParams*sizeof(*transitionMatrix));
  ALERT(cudaStat, "transitionMatrix device memory allocation failed");

  cudaStat = hipMalloc ((void**)&transitionMatrix_temp_devPtr, dynamParams*dynamParams*sizeof(*transitionMatrix_temp));
  ALERT(cudaStat, "transitionMatrix device memory allocation failed");

  cudaStat = hipMalloc ((void**)&processNoiseCov_devPtr, dynamParams*dynamParams*sizeof(*processNoiseCov));
  ALERT(cudaStat, "processNoiseCov device memory allocation failed");

  cudaStat = hipMalloc ((void**)&measurementMatrix_devPtr, measureParams*dynamParams*sizeof(*measurementMatrix));
  ALERT(cudaStat, "measurementMatrix device memory allocation failed");

  cudaStat = hipMalloc ((void**)&measurementNoiseCov_devPtr, measureParams*measureParams*sizeof(*measurementNoiseCov));
  ALERT(cudaStat, "measurementNoiseCov device memory allocation failed");

  cudaStat = hipMalloc ((void**)&measurement_devPtr, measureParams*sizeof(*measurement));
  ALERT(cudaStat, "measurement device memory allocation failed");

  cudaStat = hipMalloc ((void**)&errorCovPre_devPtr, dynamParams*dynamParams*sizeof(*errorCovPre));
  ALERT(cudaStat, "errorCovPre device memory allocation failed");

  cudaStat = hipMalloc ((void**)&errorCovPost_devPtr, dynamParams*dynamParams*sizeof(*errorCovPost));
  ALERT(cudaStat, "errorCovPost device memory allocation failed");

  cudaStat = hipMalloc ((void**)&gain_devPtr, dynamParams*measureParams*sizeof(*gain));
  ALERT(cudaStat, "gain device memory allocation failed");

  cudaStat = hipMalloc ((void**)&temp1_devPtr, dynamParams*dynamParams*sizeof(*temp1));
  ALERT(cudaStat, "temp1 device memory allocation failed");

  cudaStat = hipMalloc ((void**)&temp2_devPtr, measureParams*dynamParams*sizeof(*temp2));
  ALERT(cudaStat, "temp2 device memory allocation failed");

  cudaStat = hipMalloc ((void**)&temp2_temp_devPtr, measureParams*dynamParams*sizeof(*temp2_temp));
  ALERT(cudaStat, "temp2 device memory allocation failed");

  cudaStat = hipMalloc ((void**)&temp3_devPtr, measureParams*measureParams*sizeof(*temp3));
  ALERT(cudaStat, "temp3 device memory allocation failed");

  cudaStat = hipMalloc ((void**)&temp4_devPtr, measureParams*dynamParams*sizeof(*temp4));
  ALERT(cudaStat, "temp4 device memory allocation failed");

  cudaStat = hipMalloc ((void**)&temp5_devPtr, measureParams*sizeof(*temp5));
  ALERT(cudaStat, "temp5 device memory allocation failed");


  if( controlParams > 0 )
    {
    cudaStat = hipMalloc ((void**)&controlMatrix_devPtr, dynamParams*controlParams*sizeof(*controlMatrix));
    ALERT(cudaStat, "controlMatrix device memory allocation failed");

    cudaStat = hipMalloc ((void**)&controlVector_devPtr, controlParams*sizeof(*controlVector));
    ALERT(cudaStat, "controlVector device memory allocation failed");
    }
  std::cout << "Creation of Device CUDA Matrices successful" << std::endl;

  set_identity(dynamParams, measureParams, controlParams);
  }

__host__ int C_cudaKalman::set_identity(int dynamParams, int measureParams, int controlParams)
  {
  std::cout << "Setting Device CUDA Matrices identity" << std::endl;

  for(int i=0;i<dynamParams*dynamParams;i++)
    {
    if(i%dynamParams+1==0 | i == 0)
      {
      this->transitionMatrix[i] = 1.0;
      this->processNoiseCov[i] = 1.0;
      }
    else
      {
      this->transitionMatrix[i] = 0.0;
      this->processNoiseCov[i] = 0.0;
      }
    }

  for(int i=0;i<measureParams*measureParams;i++)
    {
    if(i%measureParams+1==0 | i == 0)
      {
      this->measurementNoiseCov[i] = 1.0;
      }
    else
      {
      this->measurementNoiseCov[i] = 0.0;
      }
    }

  for(int i=0;i<measureParams*dynamParams;i++)
    {
    if(i%measureParams+1==0 | i == 0)
      {
      this->measurementMatrix[i] = 1.0;
      }
    else
      {
      this->measurementMatrix[i] = 0.0;
      }
    }

  for(int i=0;i<dynamParams*measureParams;i++)
    {
    if(i%dynamParams+1==0 | i == 0)
      {
      this->gain[i] = 1.0;
      }
    else
      {
      this->gain[i] = 0.0;
      }
    }

  stat =  hipblasSetMatrix(dynamParams, dynamParams, sizeof(float), transitionMatrix, dynamParams, transitionMatrix_devPtr, dynamParams);
  ALERT(stat, "hipblasSetMatrix transitionMatrix failed");

  stat =  hipblasSetMatrix(dynamParams, dynamParams, sizeof(float), processNoiseCov, dynamParams, processNoiseCov_devPtr, dynamParams);
  ALERT(stat, "hipblasSetMatrix processNoiseCov failed");

  stat =  hipblasSetMatrix(measureParams, measureParams, sizeof(float), measurementNoiseCov, measureParams, measurementNoiseCov_devPtr, measureParams);
  ALERT(stat, "hipblasSetMatrix measurementNoiseCov failed");

  stat =  hipblasSetMatrix(measureParams, dynamParams, sizeof(float), measurementMatrix, measureParams, measurementMatrix_devPtr, measureParams);
  ALERT(stat, "hipblasSetMatrix measurementMatrix failed");

  stat =  hipblasSetMatrix(dynamParams, measureParams, sizeof(float), gain, dynamParams, gain_devPtr, dynamParams);
  ALERT(stat, "hipblasSetMatrix gain failed");

  if(controlParams > 0)
    {
    for(int i = 0; i < controlParams; i++)
      {
      this->controlVector[i] = -9.807f;
      }
    stat =  hipblasSetVector(controlParams, sizeof(float), controlVector, 1, controlVector_devPtr, 1);
    ALERT(stat, "hipblasSetMatrix gain failed");


    }
  std::cout << "Setting Device CUDA Matrices identity successful" << std::endl;

  }
bool C_cudaKalman::deleteMatrix()
  {
  delete [] this->statePre;
  delete [] this->statePost;

  }
__device__ void C_cudaKalman::correct            ()
  {
  const float* alpha = new float {1.0f};
  const float* beta = new float {0.0f};
  // temp2 = H*P'(k)
  // temp2 = measurementMatrix * errorCovPre;
  hipblasSgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N,measureParams,measureParams,measureParams,alpha, this->measurementMatrix_devPtr, measureParams, errorCovPre_devPtr, dynamParams, beta, this->temp2_devPtr, measureParams);

  // temp2_temp = temp2*Ht
  //gemm(temp2, measurementMatrix, 1, measurementNoiseCov, 1, temp3, GEMM_2_T);
  // Calculate: c = (alpha*a) * b + (beta*c)
  // MxN = MxK * KxN
  // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb, beta, C, ldc
  hipblasSgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_T,measureParams, dynamParams, measureParams, alpha, temp2_devPtr, measureParams, measurementMatrix_devPtr, measureParams, beta, temp2_temp_devPtr, measureParams);

  // temp2 = temp2_temp + R
  hipblasSgeam(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measureParams, measureParams, alpha, this->temp2_temp_devPtr, measureParams, alpha, this->measurementNoiseCov_devPtr, measureParams, this->temp2_devPtr, measureParams);



  }
__device__ void C_cudaKalman::predict            ()
  {
  //CUDA MEMCOPY ALPHA BETA EMPTY
  const float* alpha = new float {1.0f};
  const float* beta = new float {0.0f};
//  This function performs the matrix-matrix multiplication
//  C = ? op ( A ) op ( B ) + ? C
//  where ? and ? are scalars, and A , B and C are matrices stored in column-major format with dimensions op ( A ) m � k , op ( B ) k � n and C m � n , respectively. Also, for matrix A
//  op ( A ) = A if  transa == HIPBLAS_OP_N A T if  transa == HIPBLAS_OP_T A H if  transa == HIPBLAS_OP_C
//  and op ( B ) is defined similarly for matrix B .


//  This function performs the matrix-vector multiplication
//  y = ? op ( A ) x + ? y
//  where A is a m � n matrix stored in column-major format, x and y are vectors, and ? and ? are scalars. Also, for matrix A
//   op ( A ) = A  if transa == HIPBLAS_OP_N A T  if transa == HIPBLAS_OP_T A H  if transa == CUBLAS_OP_H

  //update the state: x'(k) = A*x(k)
  //statePre = transitionMatrix*statePost;
  hipblasSgemv(this->handle, HIPBLAS_OP_N, dynamParams, dynamParams, alpha, this->transitionMatrix_devPtr, dynamParams, statePost_devPtr, 1, beta, this->statePre_devPtr, 1);
  if(this->controlParams > 0)
    {
    //NICHT GEBRAUCHT F�R STACHNISS MODELL
    // update error covariance matrices: temp1 = A*P(k)
    // x'(k) = x'(k) + B*u(k)
    //statePre += controlMatrix*control;
    __syncthreads();
    hipblasSgemv(this->handle, HIPBLAS_OP_N, dynamParams, controlParams, alpha, this->controlMatrix_devPtr, dynamParams, controlMatrix_devPtr, 1, alpha, this->statePre_devPtr, 1);
    }

  // Calculate: c = (alpha*a) * b + (beta*c)
  // MxN = MxK * KxN
  // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb, beta, C, ldc
  // update error covariance matrices: temp1 = A*P(k)
  //temp1 = transitionMatrix*errorCovPost;
  hipblasSgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N,dynamParams,dynamParams,dynamParams,alpha, this->transitionMatrix_devPtr, dynamParams, errorCovPost_devPtr, dynamParams, beta, this->temp1_devPtr, dynamParams);

  // A_temp = temp1*At
  //gemm(temp1, transitionMatrix, 1, processNoiseCov, 1, errorCovPre, GEMM_2_T);
  hipblasSgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_T, dynamParams, dynamParams, dynamParams, alpha, this->temp1_devPtr, dynamParams, this->transitionMatrix_devPtr, dynamParams, beta ,this->transitionMatrix_temp_devPtr, dynamParams);

  // P'(k) = A_temp + Q
  hipblasSgeam(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dynamParams, dynamParams, alpha, this->transitionMatrix_temp_devPtr, dynamParams, alpha, processNoiseCov_devPtr, dynamParams, errorCovPre_devPtr, dynamParams);

  // handle the case when there will be measurement before the next predict.
  //statePre.copyTo(statePost);
  //errorCovPre.copyTo(errorCovPost);
  hipblasScopy(this->handle, dynamParams, this->statePre_devPtr, 1, this->statePost_devPtr,1);
  hipblasScopy(this->handle, dynamParams, this->errorCovPre_devPtr, 1, this->errorCovPost_devPtr,1);

  }
void C_cudaKalman::firstMeasurement   ()
  {

  }
