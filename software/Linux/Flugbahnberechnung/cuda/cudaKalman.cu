#include "cudaKalman.cuh"
#include <stdio.h>
#include <iostream>


using namespace cudaKalman;

C_cudaKalman::C_cudaKalman()
  {
  init();
  initMatrix(9,3,0);
  }
C_cudaKalman::~C_cudaKalman()
  {

  }
void C_cudaKalman::init               ()
  {
  std::cout << "hello from inti" << std::endl;
  hipblasCreate(&this->handle);
  hipblasSetStream(this->handle, this->streamkalman);

  }
void C_cudaKalman::deinit             ()
  {
  hipblasDestroy(this->handle);

  }

bool C_cudaKalman::initMatrix         (int dynamParams, int measureParams, int controlParams)
  {
  //Erzeuge neue Arrays
  std::cout << "Creating Host CUDA Matrices" << std::endl;
  this->statePre= new float[dynamParams];
  this->statePost= new float[dynamParams];
  for(int i=0;i<dynamParams;i++)
    {
    this->statePre[i]=0.0;
    this->statePost[i]=0.0;
    }

  this->transitionMatrix = new float[dynamParams*dynamParams];
  this->processNoiseCov = new float[dynamParams*dynamParams];

  this->measurementMatrix = new float[measureParams*dynamParams];
  for(int i=0;i<measureParams*dynamParams;i++)
    {
    this->measurementMatrix[i]=0.0;
    }

  this->measurementNoiseCov = new float[measureParams*measureParams];
  for(int i=0;i<measureParams*measureParams;i++)
    {
    this->measurementNoiseCov[i]=0.0;
    }

  this->measurement = new float[measureParams];
  for(int i=0;i<measureParams;i++)
    {
    this->measurement[i]=0.0;
    }


  this->errorCovPre = new float[dynamParams*dynamParams];
  this->errorCovPost = new float[dynamParams*dynamParams];
  for(int i=0;i<dynamParams*dynamParams;i++)
    {
    this->errorCovPre[i]=0.0;
    this->errorCovPost[i]=0.0;
    this->transitionMatrix[i]=0.0;
    this->processNoiseCov[i]=0.0;
    }


  this->gain = new float[dynamParams*measureParams];
  if( controlParams > 0 )
    {
    this->controlMatrix = new float[dynamParams*controlParams];
    this->controlVector = new float[controlParams];
    }

  if (!statePre | !statePost |!transitionMatrix | !processNoiseCov |!measurementMatrix | !measurementNoiseCov |!measurement | !errorCovPre |!errorCovPost)
    {
    return false;
    }
  std::cout << "Creation of Host CUDA Matrices successful" << std::endl;


  //Schreibe die Initialwerte in die jeweiligen Matrizen
  std::cout << "Creating Device CUDA Matrices" << std::endl;

  cudaStat = hipMalloc ((void**)&statePre_devPtr, dynamParams*sizeof(*statePre));
  if (cudaStat != hipSuccess)
    {
    printf ("statePre device memory allocation failed");
    return EXIT_FAILURE;
    }
  cudaStat = hipMalloc ((void**)&statePost_devPtr, dynamParams*sizeof(*statePost));
  if (cudaStat != hipSuccess)
    {
    printf ("statePost device memory allocation failed");
    return EXIT_FAILURE;
    }

  cudaStat = hipMalloc ((void**)&transitionMatrix_devPtr, dynamParams*dynamParams*sizeof(*transitionMatrix));
  if (cudaStat != hipSuccess)
    {
    printf ("statePre device memory allocation failed");
    return EXIT_FAILURE;
    }
  cudaStat = hipMalloc ((void**)&processNoiseCov_devPtr, dynamParams*dynamParams*sizeof(*processNoiseCov));
  if (cudaStat != hipSuccess)
    {
    printf ("statePost device memory allocation failed");
    return EXIT_FAILURE;
    }

  cudaStat = hipMalloc ((void**)&measurementMatrix_devPtr, measureParams*dynamParams*sizeof(*measurementMatrix));
  if (cudaStat != hipSuccess)
    {
    printf ("statePre device memory allocation failed");
    return EXIT_FAILURE;
    }
  cudaStat = hipMalloc ((void**)&measurementNoiseCov_devPtr, measureParams*measureParams*sizeof(*measurementNoiseCov));
  if (cudaStat != hipSuccess)
    {
    printf ("statePost device memory allocation failed");
    return EXIT_FAILURE;
    }
  cudaStat = hipMalloc ((void**)&measurement_devPtr, measureParams*sizeof(*measurement));
  if (cudaStat != hipSuccess)
    {
    printf ("statePost device memory allocation failed");
    return EXIT_FAILURE;
    }

  cudaStat = hipMalloc ((void**)&errorCovPre_devPtr, dynamParams*dynamParams*sizeof(*errorCovPre));
  if (cudaStat != hipSuccess)
    {
    printf ("statePre device memory allocation failed");
    return EXIT_FAILURE;
    }
  cudaStat = hipMalloc ((void**)&errorCovPost_devPtr, dynamParams*dynamParams*sizeof(*errorCovPost));
  if (cudaStat != hipSuccess)
    {
    printf ("statePost device memory allocation failed");
    return EXIT_FAILURE;
    }

  cudaStat = hipMalloc ((void**)&gain_devPtr, dynamParams*measureParams*sizeof(*gain));
  if (cudaStat != hipSuccess)
    {
    printf ("statePost device memory allocation failed");
    return EXIT_FAILURE;
    }


  if( controlParams > 0 )
    {
    cudaStat = hipMalloc ((void**)&controlMatrix_devPtr, dynamParams*controlParams*sizeof(*controlMatrix));
    if (cudaStat != hipSuccess)
      {
      printf ("statePre device memory allocation failed");
      return EXIT_FAILURE;
      }
    cudaStat = hipMalloc ((void**)&controlVector_devPtr, controlParams*sizeof(*controlVector));
    if (cudaStat != hipSuccess)
      {
      printf ("statePost device memory allocation failed");
      return EXIT_FAILURE;
      }
    }
  std::cout << "Creation of Device CUDA Matrices successful" << std::endl;

  set_identity(dynamParams, measureParams, controlParams);
  }

void C_cudaKalman::set_identity(int dynamParams, int measureParams, int controlParams)
  {
  for(int i=0;i<dynamParams*dynamParams;i++)
    {
    if(i%dynamParams+1==0)
      {
      this->transitionMatrix[i] = 1.0;
      this->processNoiseCov[i] = 1.0;
      }
    else
      {
      this->transitionMatrix[i] = 0.0;
      this->processNoiseCov[i] = 0.0;
      }
    }

  for(int i=0;i<measureParams*measureParams;i++)
    {
    if(i%measureParams+1==0)
      {
      this->measurementNoiseCov[i] = 1.0;
      }
    else
      {
      this->measurementNoiseCov[i] = 0.0;
      }
    }


  }
bool C_cudaKalman::deleteMatrix()
  {
  delete [] this->statePre;
  delete [] this->statePost;

  }
void C_cudaKalman::correct            ()
  {

  }
void C_cudaKalman::predict            ()
  {
  //CUDA MEMCOPY ALPHA BETA EMPTY
  float* empty = new float[1];
  const float* alpha = new float {1.0f};
  const float* beta = new float {0.0f};
//  This function performs the matrix-matrix multiplication
//  C = ? op ( A ) op ( B ) + ? C
//  where ? and ? are scalars, and A , B and C are matrices stored in column-major format with dimensions op ( A ) m � k , op ( B ) k � n and C m � n , respectively. Also, for matrix A
//  op ( A ) = A if  transa == HIPBLAS_OP_N A T if  transa == HIPBLAS_OP_T A H if  transa == HIPBLAS_OP_C
//  and op ( B ) is defined similarly for matrix B .


//  This function performs the matrix-vector multiplication
//  y = ? op ( A ) x + ? y
//  where A is a m � n matrix stored in column-major format, x and y are vectors, and ? and ? are scalars. Also, for matrix A
//   op ( A ) = A  if transa == HIPBLAS_OP_N A T  if transa == HIPBLAS_OP_T A H  if transa == CUBLAS_OP_H

  hipblasSgemv(this->handle, HIPBLAS_OP_N, dynamParams, dynamParams, alpha, this->transitionMatrix, dynamParams, statePost, 1, beta, empty, 1);



  }
void C_cudaKalman::firstMeasurement   ()
  {

  }
