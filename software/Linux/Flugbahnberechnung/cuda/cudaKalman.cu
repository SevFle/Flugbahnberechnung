#include "cudaKalman.cuh"
#include <stdio.h>
#include <iostream>


using namespace cudaKalman;

C_cudaKalman::C_cudaKalman()
  {
  init();
  initMatrix(9,3,0);
  }
C_cudaKalman::~C_cudaKalman()
  {

  }
void C_cudaKalman::init               ()
  {
  std::cout << "hello from inti" << std::endl;
  hipblasCreate(&this->handle);
  hipblasSetStream(this->handle, this->streamkalman);

  }
void C_cudaKalman::deinit             ()
  {
  hipblasDestroy(this->handle);

  }

bool C_cudaKalman::initMatrix         (int dynamParams, int measureParams, int controlParams)
  {
  //Erzeuge neue Arrays
  this->statePre= new float[dynamParams];
  this->statePost= new float[dynamParams];

  this->transitionMatrix = new float[dynamParams*dynamParams];
  this->processNoiseCov = new float[dynamParams*dynamParams];

  this->measurementMatrix = new float[measureParams*dynamParams];
  this->measurementNoiseCov = new float[measureParams*measureParams];
  this->measurement = new float[measureParams];

  this->errorCovPre = new float[dynamParams*dynamParams];
  this->errorCovPost = new float[dynamParams*dynamParams];

  this->gain = new float[dynamParams*measureParams];
  if( controlParams > 0 )
    {
    this->controlMatrix = new float[dynamParams*controlParams];
    this->controlVector = new float[controlParams];
    }

  if (!statePre | !statePost |!transitionMatrix | !processNoiseCov |!measurementMatrix | !measurementNoiseCov |!measurement | !errorCovPre |!errorCovPost)
    {
    return false;
    }

  //Schreibe die Initialwerte in die jeweiligen Matrizen



//  for (j = 0; j < dynamParams; j++)
//    {
//    a[IDX2C(i,j,M)] = (float)(i * M + j + 1);
//    }

//  cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
//  if (cudaStat != hipSuccess) {
//      printf ("device memory allocation failed");
//      return EXIT_FAILURE;
//  }

  }
bool C_cudaKalman::deleteMatrix()
  {
  delete [] this->statePre;
  delete [] this->statePost;

  }
