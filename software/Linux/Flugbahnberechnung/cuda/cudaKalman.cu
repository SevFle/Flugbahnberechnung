#include "hip/hip_runtime.h"
#include "cudaKalman.cuh"
#include <stdio.h>
#include <iostream>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

#define ALERT(cudaStat, message) \
  if (cudaStat != hipSuccess){ \
    printf("\n");\
    printf(message);\
    printf("\n");\
    return EXIT_FAILURE;\
    }
#define PERR(call) \
  if (call) {\
   fprintf(stderr, "%s:%d Error [%s] on "#call"\n", __FILE__, __LINE__,\
      hipGetErrorString(hipGetLastError()));\
   exit(1);\
  }
#define ERRCHECK \
  if (hipPeekAtLastError()) { \
    fprintf(stderr, "%s:%d Error [%s]\n", __FILE__, __LINE__,\
       hipGetErrorString(hipGetLastError()));\
    exit(1);\
  }
using namespace cudaKalman;
__global__ void MatSubt                 (float* A,    float* B,   float* C)
  {
  int x = threadIdx.x;
  int y = threadIdx.y;
  int id = blockDim.x*y+x;
  C[id]=A[id]-B[id];
  }
__global__ void inv_kernel              (float *a_i,  float *c_o, int n)
{
  int *p = (int *)malloc(3*sizeof(int));
  int *info = (int *)malloc(sizeof(int));
  int batch;
  hipblasHandle_t hdl;
  hipblasStatus_t status = hipblasCreate(&hdl);
  printf("handle %d n = %d\n", status, n);

  info[0] = 0;
  batch = 1;
  float **a = (float **)malloc(sizeof(float *));
  *a = a_i;
  const float **aconst = (const float **)a;
  float **c = (float **)malloc(sizeof(float *));
  *c = c_o;
  // See
  // http://docs.nvidia.com/cuda/pdf/CUDA_Dynamic_Parallelism_Programming_Guide.pdf
  //http://stackoverflow.com/questions/27094612/cublas-matrix-inversion-from-device
  status = hipblasSgetrfBatched(hdl, n, a, n, p, info, batch);
  __syncthreads();
  printf("rf %d info %d\n", status, info[0]);
  status = hipblasSgetriBatched(hdl, n, aconst, n, p,
      c, n, info, batch);
  __syncthreads();
  printf("ri %d info %d\n", status, info[0]);
  hipblasDestroy(hdl);
  printf("done\n");
}

C_cudaKalman::C_cudaKalman              ()
  {
  init();
  this->initMatrix(9,3,0);
  this->set_identity(9, 3, 0);

  }
C_cudaKalman::~C_cudaKalman             ()
  {
  deinit();
  deleteMatrix();
  }
void  C_cudaKalman::init                ()
  {
  hipblasCreate(&this->handle);
  hipblasSetStream(this->handle, this->streamkalman);
  }
void  C_cudaKalman::deinit              ()
  {
  hipblasDestroy(this->handle);

  }
bool  C_cudaKalman::initMatrix          (int dynamParams, int measureParams, int controlParams)
  {
  this->dynamParams = dynamParams;
  this->measureParams = measureParams;
  this->controlParams = controlParams;
  //Erzeuge neue Arrays
  std::cout << "Creating Host CUDA Matrices" << std::endl;
//  a = (float *)malloc (M * N * sizeof (*a));
//  if (!a) {
//      printf ("host memory allocation failed");
//      return EXIT_FAILURE;
//  }
//  for (j = 0; j < N; j++) {
//      for (i = 0; i < M; i++) {
//          a[IDX2C(i,j,M)] = (float)(i * M + j + 1);
//      }
//  }
                                                //ROWS * COLS * SIZEOF(FLOAT)
  this->statePre                = (float*)malloc(1*dynamParams*sizeof(this->statePre));
  this->statePost               = (float*)malloc(1*dynamParams*sizeof(this->statePre));
  for (int j = 1; j <= dynamParams; j++)
    {
    for (int i = 1; i <= 1; i++)
      {
      this->statePre[IDX2C(i,j,1)]                = (float)(i * 1 + j + 1);
      this->statePost[IDX2C(i,j,1)]               = (float)(i * 1 + j + 1);
      }
    }
  for (int j = 1; j <= dynamParams; j++)
    {
    for (int i = 1; i <= 1; i++)
      {
      this->statePre[IDX2C(i,j,1)]                = 0.0;
      this->statePre[IDX2C(i,j,1)]                = 0.0;
      }
    }
                                                //ROWS * COLS * SIZEOF(FLOAT)
  this->transitionMatrix        = (float*)malloc(dynamParams*dynamParams*sizeof(this->transitionMatrix));
  this->processNoiseCov         = (float*)malloc(dynamParams*dynamParams*sizeof(this->processNoiseCov));
  this->errorCovPre             = (float*)malloc(dynamParams*dynamParams*sizeof(this->errorCovPre));
  this->errorCovPost            = (float*)malloc(dynamParams*dynamParams*sizeof(this->errorCovPost));
  for (int j = 1; j <= dynamParams; j++)
    {
    for (int i = 1; i <= dynamParams; i++)
      {
      this->transitionMatrix[IDX2C(i,j,dynamParams)]        = (float)(i * dynamParams + j + 1);
      this->processNoiseCov[IDX2C(i,j,dynamParams)]         = (float)(i * dynamParams + j + 1);
      this->errorCovPre[IDX2C(i,j,dynamParams)]             = (float)(i * dynamParams + j + 1);
      this->errorCovPost[IDX2C(i,j,dynamParams)]            = (float)(i * dynamParams + j + 1);
      }
    }
  for (int j = 1; j <= dynamParams; j++)
    {
    for (int i = 1; i <= dynamParams; i++)
      {
        this->transitionMatrix[IDX2C(i,j,dynamParams)]        = 0.0;
        this->processNoiseCov[IDX2C(i,j,dynamParams)]         = 0.0;
        this->errorCovPre[IDX2C(i,j,dynamParams)]             = 0.0;
        this->errorCovPost[IDX2C(i,j,dynamParams)]            = 0.0;
      }
    }
                                                //ROWS * COLS * SIZEOF(FLOAT)
  this->measurementMatrix       = (float*)malloc(measureParams*dynamParams*sizeof(this->measurementMatrix));
  for (int j = 1; j <= dynamParams; j++)
    {
    for (int i = 1; i <= measureParams; i++)
      {
      this->measurementMatrix[IDX2C(i,j,measureParams)]       = (float)(i * measureParams + j + 1);
      }
    }
  for (int j = 1; j <= dynamParams; j++)
    {
    for (int i = 1; i <= measureParams; i++)
      {
        this->measurementMatrix[IDX2C(i,j,measureParams)]         = 0.0;
      }
    }
                                                //ROWS * COLS * SIZEOF(FLOAT)
  this->measurementNoiseCov     = (float*)malloc(measureParams*measureParams*sizeof(this->measurementMatrix));
  for (int j = 1; j <= measureParams; j++)
    {
    for (int i = 1; i <= measureParams; i++)
      {
      this->measurementNoiseCov[IDX2C(i,j,measureParams)]       = (float)(i * measureParams + j + 1);
      }
    }
  for (int j = 1; j <= measureParams; j++)
    {
    for (int i = 1; i <= measureParams; i++)
      {
        this->measurementNoiseCov[IDX2C(i,j,measureParams)]         = 0.0;
      }
    }
                                                //ROWS * COLS * SIZEOF(FLOAT)
  this->measurement             = (float*)malloc(1*measureParams*sizeof(this->measurement));
  for (int j = 1; j <= measureParams; j++)
    {
    for (int i = 1; i <= 1; i++)
      {
      this->measurement[IDX2C(i,j,1)]           = (float)(i * 1 + j + 1);
      }
    }
  for (int j = 1; j <= measureParams; j++)
    {
    for (int i = 1; i <= 1; i++)
      {
      this->measurement[IDX2C(i,j,1)]         = 0.0;
      }
    }
                                                //ROWS * COLS * SIZEOF(FLOAT)
  this->gain                   = (float*)malloc(dynamParams*measureParams*sizeof(this->gain));
  for (int j = 1; j <= measureParams; j++)
    {
    for (int i = 1; i <= dynamParams; i++)
      {
      this->measurementNoiseCov[IDX2C(i,j,dynamParams)]       = (float)(i * dynamParams + j + 1);
      }
    }
  for (int j = 1; j <= measureParams; j++)
    {
    for (int i = 1; i <= dynamParams; i++)
      {
        this->measurementNoiseCov[IDX2C(i,j,dynamParams)]      = 0.0;
      }
    }

  if( controlParams > 0 )
    {
    this->controlMatrix = new float[dynamParams*controlParams];
    this->controlVector = new float[controlParams];
    }

  if (!statePre | !statePost |!transitionMatrix | !processNoiseCov |!measurementMatrix | !measurementNoiseCov |!measurement | !errorCovPre |!errorCovPost)
    {
    return false;
    }
  std::cout << "Creation of Host CUDA Matrices successful" << std::endl;


  //Schreibe die Initialwerte in die jeweiligen Matrizen
  std::cout << "Creating Device CUDA Matrices" << std::endl;

  cudaStat = hipMalloc ((void**)&statePre_devPtr, dynamParams*sizeof(*statePre));
  ALERT(cudaStat, "statePre device memory allocation failed");
  this->print_matrix(statePre, dynamParams, 1, "statePre");

  cudaStat = hipMalloc ((void**)&statePost_devPtr, dynamParams*sizeof(*statePost));
  ALERT(cudaStat, "statePost device memory allocation failed");
  this->print_matrix(statePost, dynamParams, 1, "statePost");

  cudaStat = hipMalloc ((void**)&transitionMatrix_devPtr, dynamParams*dynamParams*sizeof(*transitionMatrix));
  ALERT(cudaStat, "transitionMatrix device memory allocation failed");
  this->print_matrix(transitionMatrix, dynamParams, dynamParams, "transitionMatrix");

  cudaStat = hipMalloc ((void**)&processNoiseCov_devPtr, dynamParams*dynamParams*sizeof(*processNoiseCov));
  ALERT(cudaStat, "processNoiseCov device memory allocation failed");
  this->print_matrix(processNoiseCov, dynamParams, dynamParams, "processNoiseCov");

  cudaStat = hipMalloc ((void**)&measurementMatrix_devPtr, measureParams*dynamParams*sizeof(*measurementMatrix));
  ALERT(cudaStat, "measurementMatrix device memory allocation failed");
  this->print_matrix(measurementMatrix, measureParams, dynamParams, "measurementMatrix");

  cudaStat = hipMalloc ((void**)&measurementNoiseCov_devPtr, measureParams*measureParams*sizeof(*measurementNoiseCov));
  ALERT(cudaStat, "measurementNoiseCov device memory allocation failed");
  this->print_matrix(measurementNoiseCov, measureParams, measureParams, "measurementNoiseCov");

  cudaStat = hipMalloc ((void**)&measurement_devPtr, measureParams*sizeof(*measurement));
  ALERT(cudaStat, "measurement device memory allocation failed");
  this->print_matrix(measurement, measureParams, 1, "measurement");

  cudaStat = hipMalloc ((void**)&errorCovPre_devPtr, dynamParams*dynamParams*sizeof(*errorCovPre));
  ALERT(cudaStat, "errorCovPre device memory allocation failed");
  this->print_matrix(errorCovPre, dynamParams, dynamParams, "errorCovPre");

  cudaStat = hipMalloc ((void**)&errorCovPost_devPtr, dynamParams*dynamParams*sizeof(*errorCovPost));
  ALERT(cudaStat, "errorCovPost device memory allocation failed");
  this->print_matrix(errorCovPost, dynamParams, dynamParams, "errorCovPost");

  cudaStat = hipMalloc ((void**)&gain_devPtr, dynamParams*measureParams*sizeof(*gain));
  ALERT(cudaStat, "gain device memory allocation failed");
  this->print_matrix(gain, dynamParams, measureParams, "gain");

  cudaStat = hipMalloc ((void**)&temp1_devPtr, dynamParams*dynamParams*sizeof(float));
  ALERT(cudaStat, "temp1 device memory allocation failed");
  this->print_matrix(temp1, dynamParams, measureParams, "temp1");

  cudaStat = hipMalloc ((void**)&temp2_devPtr, measureParams*dynamParams*sizeof(float));
  ALERT(cudaStat, "temp2 device memory allocation failed");
  this->print_matrix(temp2, measureParams, measureParams, "temp2");

  cudaStat = hipMalloc ((void**)&temp3_devPtr, measureParams*measureParams*sizeof(float));
  ALERT(cudaStat, "temp3 device memory allocation failed");
  this->print_matrix(temp3, measureParams, measureParams, "temp3");

  cudaStat = hipMalloc ((void**)&temp3_inv_devPtr, measureParams*measureParams*sizeof(float));
  ALERT(cudaStat, "temp3_inv device memory allocation failed");
  this->print_matrix(temp3_inv_devPtr, measureParams, measureParams, "temp3");



  cudaStat = hipMalloc ((void**)&temp4_devPtr, measureParams*dynamParams*sizeof(*temp4));
  ALERT(cudaStat, "temp4 device memory allocation failed");
  this->print_matrix(temp4, measureParams, dynamParams, "temp4");

  cudaStat = hipMalloc ((void**)&temp5_devPtr, measureParams*sizeof(*temp5));
  ALERT(cudaStat, "temp5 device memory allocation failed");
  this->print_matrix(temp5, measureParams, 1, "temp5");




  if( controlParams !=0 )
    {
    cudaStat = hipMalloc ((void**)&controlMatrix_devPtr, dynamParams*controlParams*sizeof(*controlMatrix));
    ALERT(cudaStat, "controlMatrix device memory allocation failed");

    cudaStat = hipMalloc ((void**)&controlVector_devPtr, controlParams*sizeof(*controlVector));
    ALERT(cudaStat, "controlVector device memory allocation failed");
    }
  std::cout << "Creation of Device CUDA Matrices successful" << std::endl;

  }

int   C_cudaKalman::set_identity        (int dynamParams, int measureParams, int controlParams)
  {
  std::cout << "Setting Device CUDA Matrices identity" << std::endl;

  for(int i=0;i<dynamParams*dynamParams;i++)
    {
    if(i%dynamParams+1==0 | i == 0)
      {
      this->transitionMatrix[i] = 1.0;
      this->processNoiseCov[i] = 1.0;
      }
    else
      {
      this->transitionMatrix[i] = 0.0;
      this->processNoiseCov[i] = 0.0;
      }
    }
  this->print_matrix(transitionMatrix, dynamParams, dynamParams, "transitionMatrix");
  this->print_matrix(processNoiseCov, dynamParams, dynamParams, "processNoiseCov");

  for(int i=0;i<measureParams*measureParams;i++)
    {
    if(i%measureParams+1==0 | i == 0)
      {
      this->measurementNoiseCov[i] = 1.0;
      }
    else
      {
      this->measurementNoiseCov[i] = 0.0;
      }
    }
  this->print_matrix(measurementNoiseCov, measureParams, measureParams, "measurementNoiseCov");

  for(int i=0;i<measureParams*dynamParams;i++)
    {
    if(i%measureParams+1==0 | i == 0)
      {
      this->measurementMatrix[i] = 1.0;
      }
    else
      {
      this->measurementMatrix[i] = 0.0;
      }
    }
  this->print_matrix(measurementMatrix, measureParams, dynamParams, "measurementMatrix");

  for(int i=0;i<dynamParams*measureParams;i++)
    {
    if(i%dynamParams+1==0 | i == 0)
      {
      this->gain[i] = 1.0;
      }
    else
      {
      this->gain[i] = 0.0;
      }
    }
  this->print_matrix(gain, dynamParams, measureParams, "gain");

  stat =  hipblasSetMatrix(dynamParams, dynamParams, sizeof(float), transitionMatrix, dynamParams, transitionMatrix_devPtr, dynamParams);
  ALERT(stat, "hipblasSetMatrix transitionMatrix failed");

  stat =  hipblasSetMatrix(dynamParams, dynamParams, sizeof(float), processNoiseCov, dynamParams, processNoiseCov_devPtr, dynamParams);
  ALERT(stat, "hipblasSetMatrix processNoiseCov failed");

  stat =  hipblasSetMatrix(measureParams, measureParams, sizeof(float), measurementNoiseCov, measureParams, measurementNoiseCov_devPtr, measureParams);
  ALERT(stat, "hipblasSetMatrix measurementNoiseCov failed");

  stat =  hipblasSetMatrix(measureParams, dynamParams, sizeof(float), measurementMatrix, measureParams, measurementMatrix_devPtr, measureParams);
  ALERT(stat, "hipblasSetMatrix measurementMatrix failed");

  stat =  hipblasSetMatrix(dynamParams, measureParams, sizeof(float), gain, dynamParams, gain_devPtr, dynamParams);
  ALERT(stat, "hipblasSetMatrix gain failed");

  if(controlParams > 0)
    {
    for(int i = 0; i < controlParams; i++)
      {
      this->controlVector[i] = -9.807f;
      }
    stat =  hipblasSetVector(controlParams, sizeof(float), controlVector, 1, controlVector_devPtr, 1);
    ALERT(stat, "hipblasSetMatrix gain failed");
    }
  else
    {
    std::cout << "Setting Device CUDA Matrices identity successful" << std::endl;
    }

  }
bool  C_cudaKalman::deleteMatrix        ()
  {
  free(this->statePre);
  free(this->statePost);
  free(this->transitionMatrix);
  free(this->processNoiseCov);
  free(this->errorCovPre);
  free(this->errorCovPost);
  free(this->measurementMatrix);
  free(this->measurementNoiseCov);
  free(this->measurement);
  free(this->gain);

  hipFree(this->statePre_devPtr);
  hipFree(this->statePost_devPtr);
  hipFree(this->transitionMatrix_devPtr);
  hipFree(this->processNoiseCov_devPtr);
  hipFree(this->temp1_devPtr);
  hipFree(this->errorCovPre_devPtr);
  hipFree(this->errorCovPost_devPtr);
  hipFree(this->measurementMatrix_devPtr);
  hipFree(this->temp2_devPtr);
  hipFree(this->temp4_devPtr);
  hipFree(this->measurementNoiseCov_devPtr);
  hipFree(this->temp3_devPtr);
  hipFree(this->measurement_devPtr);
  hipFree(this->temp5_devPtr);
  hipFree(this->gain_devPtr);
  }
int   C_cudaKalman::d_correct           ()
  {
  const float* alpha = new float {1.0f};
  const float* beta = new float {0.0f};
  // temp2 = H*P'(k)
  // temp2 = measurementMatrix * errorCovPre;
  stat = hipblasSgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N,measureParams,measureParams,measureParams,
                        alpha, this->measurementMatrix_devPtr, measureParams, errorCovPre_devPtr, dynamParams, beta, this->temp2_devPtr, measureParams);
  ALERT(stat, "correct - temp2 = H*P'(k)");

  // temp2_temp = temp2*Ht
  //gemm(temp2, measurementMatrix, 1, measurementNoiseCov, 1, temp3, GEMM_2_T);
  // Calculate: c = (alpha*a) * b + (beta*c)
  // MxN = MxK * KxN
  // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb, beta, C, ldc
  stat = hipblasSgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_T,measureParams, dynamParams, measureParams,
                        alpha, temp2_devPtr, measureParams, measurementMatrix_devPtr, measureParams, beta, this->temp7_devPtr, measureParams);
  ALERT(stat, "correct - temp7_devPtr = temp2*Ht");

  // temp2 = temp7_devPtr + R
  stat = hipblasSgeam(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measureParams, measureParams,
                     alpha, this->temp7_devPtr, measureParams, alpha, this->measurementNoiseCov_devPtr, measureParams, this->temp2_devPtr, measureParams);
  ALERT(stat, "correct - temp2 = temp7_devPtr + R failed");

  inv_kernel<<<1, 1>>>(temp3_devPtr, temp3_inv_devPtr, measureParams);
  // MxN = MxK * KxN
  // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb, beta, C, ldc

  stat = hipblasSgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N,measureParams, dynamParams, measureParams,
                        alpha, temp3_inv_devPtr, measureParams, temp2_devPtr, measureParams, beta, temp4_devPtr, measureParams);
  ALERT(stat, "correct - temp4 = temp3_inv * temp2");

  stat = hipblasSgeam(this->handle, HIPBLAS_OP_T, HIPBLAS_OP_N, measureParams, dynamParams,
                     alpha, temp4_devPtr, measureParams, beta, this->gain_devPtr, dynamParams, this->gain_devPtr, dynamParams);
  ALERT(stat, "correct - K(k)=temp4_transposed");


  stat = hipblasSgemm(this->handle,HIPBLAS_OP_N, HIPBLAS_OP_N,measureParams, 1, dynamParams, alpha,
                        this->measurementMatrix_devPtr, measureParams, this->statePre, dynamParams, beta, this->temp8_devPtr, measureParams);
  ALERT(stat, "correct - temp8 = H * x'(k)");

  MatSubt<<<1,measureParams>>>(measurement_devPtr, temp8_devPtr, temp5_devPtr);

  stat = hipblasSgemv(this->handle, HIPBLAS_OP_N,dynamParams, measureParams, alpha,
                        this->gain_devPtr, dynamParams, this->temp5_devPtr, 1, beta, this->temp5_devPtr, 1);
  ALERT(stat, "correct - temp9 = K(k)* temp5");

  stat = hipblasSgeam(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dynamParams, 1, alpha,
                     this->statePre, dynamParams, alpha, this->temp9_devPtr, dynamParams, this->statePost_devPtr, dynamParams);
  ALERT(stat, "correct - x(k) = x'(k) + temp9");

  stat = hipblasSgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dynamParams, measureParams, dynamParams, alpha,
                        this->gain_devPtr, dynamParams, temp2_devPtr, measureParams, beta, this->temp10_devPtr, dynamParams);
  ALERT(stat, "correct - temp10 = K(k)* temp2");

  MatSubt<<<1,dynamParams>>>(temp10_devPtr, errorCovPre_devPtr, errorCovPost_devPtr);
  }
int   C_cudaKalman::d_predict           ()
  {
  //CUDA MEMCOPY ALPHA BETA EMPTY
  const float* alpha = new float {1.0f};
  const float* beta = new float {0.0f};

//  This function performs the matrix-vector multiplication
//  y = ( A ) x + ? y
//  where A is a m � n matrix stored in column-major format, x and y are vectors, and ? and ? are scalars. Also, for matrix A
//   op ( A ) = A  if transa == HIPBLAS_OP_N A T  if transa == HIPBLAS_OP_T A H  if transa == CUBLAS_OP_H

  //update the state: x'(k) = A*x(k)
  //statePre = transitionMatrix*statePost;
  stat = hipblasSgemv(this->handle, HIPBLAS_OP_N, dynamParams, dynamParams, alpha, this->transitionMatrix_devPtr, dynamParams, statePost_devPtr, 1, beta, this->statePre_devPtr, 1);
  ALERT(stat, "update - x'(k) = A*x(k)");

  if(this->controlParams > 0)
    {
    //NICHT GEBRAUCHT F�R STACHNISS MODELL
    // update error covariance matrices: temp1 = A*P(k)
    // x'(k) = x'(k) + B*u(k)
    //statePre += controlMatrix*control;
    hipblasSgemv(this->handle, HIPBLAS_OP_N, dynamParams, controlParams, alpha, this->controlMatrix_devPtr, dynamParams, controlMatrix_devPtr, 1, alpha, this->statePre_devPtr, 1);
    
    }

  // Calculate: c = (alpha*a) * b + (beta*c)
  // MxN = MxK * KxN
  // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb, beta, C, ldc
  // update error covariance matrices: temp1 = A*P(k)
  //temp1 = transitionMatrix*errorCovPost;
  stat = hipblasSgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N,dynamParams,dynamParams,dynamParams,alpha, this->transitionMatrix_devPtr, dynamParams, errorCovPost_devPtr, dynamParams, beta, this->temp1_devPtr, dynamParams);
  ALERT(stat, "update - temp1 = A*P(k)");

  // A_temp = temp1*At
  //gemm(temp1, transitionMatrix, 1, processNoiseCov, 1, errorCovPre, GEMM_2_T);
  stat = hipblasSgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_T, dynamParams, dynamParams, dynamParams, alpha, this->temp1_devPtr, dynamParams, this->transitionMatrix_devPtr, dynamParams, beta ,this->transitionMatrix_temp_devPtr, dynamParams);
  ALERT(stat, "update - A_temp = temp1*At");

  // P'(k) = A_temp + Q
  stat = hipblasSgeam(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dynamParams, dynamParams, alpha, this->transitionMatrix_temp_devPtr, dynamParams, alpha, processNoiseCov_devPtr, dynamParams, errorCovPre_devPtr, dynamParams);
  ALERT(stat, "update - P'(k) = A_temp + Q");


  // handle the case when there will be measurement before the next predict.
  stat = hipblasScopy(this->handle, dynamParams, this->statePre_devPtr, 1, this->statePost_devPtr,1);
  ALERT(stat, "update - hipblasScopy statePre");

  stat = hipblasScopy(this->handle, dynamParams, this->errorCovPre_devPtr, 1, this->errorCovPost_devPtr,1);
  ALERT(stat, "update - hipblasScopy errorCovPre");
  }
int   C_cudaKalman::h_firstMeasurement  ()
  {

  }
void  C_cudaKalman::print_matrix        (const float *A, int nr_rows_A, int nr_cols_A, std::string Name)
  {//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
  std::cout << Name << std::endl;
  for(int i = 0; i < nr_rows_A; ++i)
    {
    for(int j = 0; j < nr_cols_A; ++j)
      {
      std::cout << A[j * nr_rows_A + i] << " ";
      }
    std::cout << std::endl;
    }
  std::cout << std::endl;
  }


