
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

//print the matrix
void matPrint(float *A, int h, int w)
{
	int i,j;
	for(i = 0;i < h;i++)
		{
		for(j = 0;j < w;j++)
			{
			printf("%f ", A[i * w + j]);
			}
		printf("\n");}
}
 

//Random Initialize
void RandomInit(float* data, int n1, int n2)
{   
	srand(1);
    for (int i = 0; i < (n1*n2); ++i)
        data[i] = (rand() % 10) + 1;
}

//Ideintity Matrix Generation
void Identity(float *data, int n)
{
	for (int i = 0; i < (n*n); i=i+1)
		{
		if((i%(n+1))==0)
			data[i] = 1;
		else
			data[i] = 0;
		}        
}






