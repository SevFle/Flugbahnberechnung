//Includes
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cutil_inline.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "matrix_kernel.cu"
#include "matrix.cu"
#include "libCuda.h"
#include <hipblas.h>




using namespace onCuda::KalmanFilter;

void allocateMemory()
  {
    //Allocate vectors  in device memory
    hipMalloc(&d_X, ns*1);
    hipMalloc(&d_P, ns*ns);
    hipMalloc(&d_F, ns*ns);
    hipMalloc(&d_Z, no*1);
    hipMalloc(&d_S, no*no);
    hipMalloc(&d_s, no*no);
    hipMalloc(&d_K, ns*no);
    hipMalloc(&d_H, no*ns);
    hipMalloc(&d_E, no*no);
    hipMalloc(&d_Ft, ns*ns);
    hipMalloc(&d_Ht, ns*no);
    hipMalloc(&d_Si, no*no);
    hipMalloc(&d_Y, no*1);
    hipMalloc(&d_I, ns*ns);
    hipMalloc(&d_Hint, no*ns);
    hipMalloc(&d_Sint, no*no);
    hipMalloc(&d_Kint, ns*no);
    hipMalloc(&d_Xint, ns*1);
    hipMalloc(&d_Pint, ns*ns);
    hipMalloc(&d_Pint2, ns*ns);
    lda = ((no+15)&~15|16);

    hipError_t ret = hipHostMalloc( (void**)&A, no*lda*sizeof(float) );
    if( ret != hipSuccess )
      {
      printf("Failed to allocate %d memory", no*lda*sizeof(float));
      return;
      }//if
    printf("\nAllocation of the Device memory completed...\n");


    for(int i = 0; i<no ; i++)
      {
      for(int j = i; j < no; j++)
        {
        A[i*lda + j] = s[i * no + j];
        A[j*lda+i] = A[i*lda+j];
        }//for
      }//for

  }//allocateMemory()

void onCuda::KalmanFilter::initialize (float *F, float *H,float *E,float *s, int ns, int no)
    //X = Estimate
    //P = Uncertainty Covariance Matrix
    //F = State transition Matrix - Prediciton Matrix
    //Z = Measurement
    //H = Measurement Function - observation model which maps the true state space into the observed space
    //E = Measurement/Observation noise
    //I = Identity Matrix

  {
  onCuda::KalmanFilter::ns = ns;
  onCuda::KalmanFilter::no = no;
  allocateMemory();

  //Init I
  memAlloc(&I,ns,ns);
  Identity(d_I, ns);
  free(&I);

  // Copy Input vectors from host memory to device memory
  hipMemcpy(d_F, F, ns*ns, hipMemcpyHostToDevice);
  hipMemcpy(d_H, H, no*ns, hipMemcpyHostToDevice);
  hipMemcpy(d_E, E, no*no, hipMemcpyHostToDevice);
  hipMemcpy(d_I, I, ns*ns, hipMemcpyHostToDevice);

  // Set the kernel arguments
  int threadsPerBlock = 256;
  int Nos = no*ns;
  int Ns =  ns;
  int No = no;
  int Ns2 = ns*ns;
  int No2 = no*no;
  int blocksPerGridNos = (Nos + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNs = (Ns + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNo = (No + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNs2 = (Ns2 + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNo2 = (No2 + threadsPerBlock - 1) / threadsPerBlock;

  //Transpose
  MatTranspose<<<blocksPerGridNos, threadsPerBlock>>>(d_Ht,d_H,no,ns);
  printf("\n Transpose of H successful\n");

  MatTranspose<<<blocksPerGridNos, threadsPerBlock>>>(d_Ft,d_F,ns,ns);
  printf("\n Transpose of F successful\n");

  //Inverse using CUBLAS

  if( hipSetDevice( dev ) != hipSuccess )
    {
    printf( "Failed to set device %d\n", dev );
    return;
    }

  if( cublasInit( ) != HIPBLAS_STATUS_SUCCESS )
    {
    printf( "failed to initialize the cublas library\n" );
    return;
    }
  printf("Cublas initialized...\n");

  invert(A, lda, no);

  for(int i = 0; i<no ; i++)
    {
    for(int j = i; j < no; j++)
      {
      si[i * no + j] = A[i * lda + j];
      si[j * no + i] = si[i * no + j];
       }
     }

  hipMemcpy(d_Si, si, no*no, hipMemcpyHostToDevice);

  }

void update()
{

  //step 1  to calculate Y = Z - HX
MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Y, d_H, d_X, no, ns);
MatSub<<<blocksPerGridNo, threadsPerBlock>>>(d_Y, d_Z, d_Y, no, 1);

//step 2 to calculate  S = HPHt + E

MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Hint, d_H, d_P, no, ns);
MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Sint, d_Hint, d_Ht, no, ns);
MatAdd<<<blocksPerGridNo2, threadsPerBlock>>>(d_S, d_Sint, d_E, no, no);

//step 3 to calcualte K = PHtSi

MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Kint, d_P, d_Ht, no, ns);

MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_K, d_Kint, d_Si, ns, no);

//step4 to calculate  X = X+ KY

MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Xint, d_K, d_Y, ns, no);
MatAdd<<<blocksPerGridNs, threadsPerBlock>>>(d_X, d_X, d_Xint, ns, 1);

//step5 to calculate [I - KH]P

MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Pint, d_K, d_H, ns, no);
MatSub<<<blocksPerGridNs2, threadsPerBlock>>>(d_Pint, d_I, d_Pint, ns, ns);
MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_Pint2, d_Pint, d_P, ns, ns);
MatCopy<<<blocksPerGridNs2, threadsPerBlock>>>(d_P, d_Pint2, ns, ns);
// Host wait for the kernel to finish
hipDeviceSynchronize();


}

float predict(float *estimate)
{
  //Prediction Phase
  // X = FX
  // P = FPFt

  //step 1 to calculate X = FX
  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_Xint, d_F, d_X, ns, ns);
  MatCopy<<<blocksPerGridNs, threadsPerBlock>>>(d_X, d_Xint, ns, 1);

  //step2 to calculate P = FPFt

  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_Pint, d_F, d_P, ns, ns);
  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_P, d_Pint, d_Ft, ns, ns);
  // Host wait for the kernel to finish
  hipDeviceSynchronize();
  // Copy result from device memory to host memory
  // h_X contains the result in host memory
  hipMemcpy(h_X, d_X, ns*1, hipMemcpyDeviceToHost);

  estimate = h_X;
}


void cleanup()
  {
  // stop and destroy timer
  hipHostFree( A );
  cublasShutdown();

  // Free host memory
  if(X)
      free(X);
  if(P)
      free(P);
  if(F)
      free(F);
  if(Ztemp)
      free(Ztemp);
  if(S)
      free(S);
  if(s)
      free(s);
  if(K)
      free(K);
  if(H)
      free(H);
  if(E)
      free(E);
  if(Ft)
      free(Ft);
  if(Ht)
      free(Ht);
  if(Si)
      free(Si);
  if(Y)
      free(Y);
  if(I)
      free(I);
  if(Hint)
      free(Hint);
  if(Sint)
      free(Sint);
  if(Kint)
      free(Kint);
  if(Xint)
      free(Xint);
  if(Pint)
      free(Pint);
  if(Pint2)
      free(Pint2);
  //printf("\n Host Cleanup Successful\n");

  // Free device memory

      if(d_X)
          free(d_X);
      if(d_P)
          free(d_P);

      if(d_F)
          free(d_F);
      if(d_Ztemp)
          free(d_Ztemp);
      if(d_S)
          free(d_S);
      if(d_s)
          free(d_s);
      if(d_K)
          free(d_K);
      if(d_H)
          free(d_H);
      if(d_E)
          free(d_E);
      if(d_Ft)
          free(d_Ft);
      if(d_Ht)
          free(d_Ht);
      if(d_Si)
          free(d_Si);
      if(d_Y)
          free(d_Y);
      if(d_I)
          free(d_I);
      if(d_Hint)
          free(d_Hint);
      if(d_Sint)
          free(d_Sint);
      if(d_Kint)
          free(d_Kint);
      if(d_Xint)
          free(d_Xint);
      if(d_Pint)
          free(d_Pint);
      if(d_Pint2)
          free(d_Pint2);

      printf("\nDevice Cleanup Successful\n");


      cutilSafeCall( hipDeviceReset() );

      exit(0);

}




