#include "hip/hip_runtime.h"
//Includes
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//#include <cutil_inline.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "matrix_kernel.cu"
#include "matrix.cu"
#include "../include/libCuda.h"
#include <hipblas.h>
#include "../src/invert.cpp"

using namespace onCuda::KalmanFilter;

C_kalman::C_kalman()
{

}

C_kalman::~C_kalman()
{

}
void C_kalman::allocateMemory()
  {
    //no == zeilen
    //ns == spalten

    //Allocate vectors  in device memory
    hipMalloc(&d_X, ns*1);
    hipMalloc(&d_P, ns*ns);
    hipMalloc(&d_F, ns*ns);
    hipMalloc(&d_Z, no*1);
    hipMalloc(&d_S, no*no);
    hipMalloc(&d_s, no*no);
    hipMalloc(&d_K, ns*no);
    hipMalloc(&d_H, no*ns);
    hipMalloc(&d_E, no*no);
    hipMalloc(&d_Ft, ns*ns);
    hipMalloc(&d_Ht, ns*no);
    hipMalloc(&d_Si, no*no);
    hipMalloc(&d_Y, no*1);
    hipMalloc(&d_I, ns*ns);
    hipMalloc(&d_Hint, no*ns);
    hipMalloc(&d_Sint, no*no);
    hipMalloc(&d_Kint, ns*no);
    hipMalloc(&d_Xint, ns*1);
    hipMalloc(&d_Pint, ns*ns);
    hipMalloc(&d_Pint2, ns*ns);
    lda = ((no+15)&~15|16);

    hipError_t ret = hipHostMalloc( (void**)&A, no*lda*sizeof(float) );
    if( ret != hipSuccess )
      {
      printf("Failed to allocate %d memory", no*lda*sizeof(float));
      return;
      }//if
    printf("\nAllocation of the Device memory completed...\n");


    for(int i = 0; i<no ; i++)
      {
      for(int j = i; j < no; j++)
        {
        A[i*lda + j] = s[i * no + j];
        A[j*lda+i] = A[i*lda+j];
        }//for
      }//for

  }//allocateMemory()

void C_kalman::initialize (float *F, float *H,float *E,float *s, int ns, int no)
    //X = Estimate
    //P = Uncertainty Covariance Matrix
    //F = State transition Matrix - Prediciton Matrix
    //Z = Measurement
    //H = Measurement Function - observation model which maps the true state space into the observed space
    //E = Measurement/Observation noise
    //I = Identity Matrix

  {
  this->ns = ns;
  this->no = no;
  allocateMemory();

  //Init I
  memAlloc(&I,ns,ns);
  Identity(d_I, ns);
  free(&I);

  // Copy Input vectors from host memory to device memory
  hipMemcpy(d_F, F, ns*ns, hipMemcpyHostToDevice);
  hipMemcpy(d_H, H, no*ns, hipMemcpyHostToDevice);
  hipMemcpy(d_E, E, no*no, hipMemcpyHostToDevice);
  hipMemcpy(d_I, I, ns*ns, hipMemcpyHostToDevice);

  // Set the kernel arguments
  int threadsPerBlock = 256;
  int Nos = no*ns;
  int Ns =  ns;
  int No = no;
  int Ns2 = ns*ns;
  int No2 = no*no;
  int blocksPerGridNos = (Nos + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNs = (Ns + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNo = (No + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNs2 = (Ns2 + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNo2 = (No2 + threadsPerBlock - 1) / threadsPerBlock;

  //Transpose
  MatTranspose<<<blocksPerGridNos, threadsPerBlock>>>(d_Ht,d_H,no,ns);
  printf("\n Transpose of H successful\n");

  MatTranspose<<<blocksPerGridNos, threadsPerBlock>>>(d_Ft,d_F,ns,ns);
  printf("\n Transpose of F successful\n");

  //Inverse using CUBLAS

  if( hipSetDevice( dev ) != hipSuccess )
    {
    printf( "Failed to set device %d\n", dev );
    return;
    }

  if( cublasInit( ) != HIPBLAS_STATUS_SUCCESS )
    {
    printf( "failed to initialize the cublas library\n" );
    return;
    }
  printf("Cublas initialized...\n");

  invert(A, lda, no);

  for(int i = 0; i<no ; i++)
    {
    for(int j = i; j < no; j++)
      {
      si[i * no + j] = A[i * lda + j];
      si[j * no + i] = si[i * no + j];
       }
     }

  hipMemcpy(d_Si, si, no*no, hipMemcpyHostToDevice);

  }

void C_kalman::correct       (float *measurement)
  {
  hipMemcpy(d_F, F, ns*ns, hipMemcpyHostToDevice);
  //step 1  to calculate Y = Z - HX
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Y, d_H, d_X, no, ns);
  MatSub<<<blocksPerGridNo, threadsPerBlock>>>(d_Y, d_Z, d_Y, no, 1);

  //step 2 to calculate  S = HPHt + E
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Hint, d_H, d_P, no, ns);
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Sint, d_Hint, d_Ht, no, ns);
  MatAdd<<<blocksPerGridNo2, threadsPerBlock>>>(d_S, d_Sint, d_E, no, no);

  //step 3 to calcualte K = PHtSi
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Kint, d_P, d_Ht, no, ns);
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_K, d_Kint, d_Si, ns, no);

  //step4 to calculate  X = X+ KY
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Xint, d_K, d_Y, ns, no);
  MatAdd<<<blocksPerGridNs, threadsPerBlock>>>(d_X, d_X, d_Xint, ns, 1);

  //step5 to calculate [I - KH]P
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Pint, d_K, d_H, ns, no);
  MatSub<<<blocksPerGridNs2, threadsPerBlock>>>(d_Pint, d_I, d_Pint, ns, ns);
  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_Pint2, d_Pint, d_P, ns, ns);
  MatCopy<<<blocksPerGridNs2, threadsPerBlock>>>(d_P, d_Pint2, ns, ns);

  // Host wait for the kernel to finish
  __syncthreads;
  hipDeviceSynchronize();
  }

float C_kalman::predict()
{
  //Prediction Phase
  // X = FX
  // P = FPFt

  //step 1 to calculate X = FX
  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_Xint, d_F, d_X, ns, ns);
  MatCopy<<<blocksPerGridNs, threadsPerBlock>>>(d_X, d_Xint, ns, 1);

  //step2 to calculate P = FPFt
  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_Pint, d_F, d_P, ns, ns);
  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_P, d_Pint, d_Ft, ns, ns);
  // Host wait for the kernel to finish
  hipDeviceSynchronize();

  // Copy result from device memory to host memory
  // h_X contains the result in host memory
  hipMemcpy(h_X, d_X, ns*1, hipMemcpyDeviceToHost);
  float estimate;
  estimate = *h_X;
  return estimate;
}


void C_kalman::cleanup()
  {
  // stop and destroy timer
  hipHostFree( A );
  cublasShutdown();

  // Free host memory
  if(X)
      free(X);
  if(P)
      free(P);
  if(F)
      free(F);
  if(Ztemp)
      free(Ztemp);
  if(S)
      free(S);
  if(s)
      free(s);
  if(K)
      free(K);
  if(H)
      free(H);
  if(E)
      free(E);
  if(Ft)
      free(Ft);
  if(Ht)
      free(Ht);
  if(Si)
      free(Si);
  if(Y)
      free(Y);
  if(I)
      free(I);
  if(Hint)
      free(Hint);
  if(Sint)
      free(Sint);
  if(Kint)
      free(Kint);
  if(Xint)
      free(Xint);
  if(Pint)
      free(Pint);
  if(Pint2)
      free(Pint2);
  //printf("\n Host Cleanup Successful\n");

  // Free device memory

  if(d_X)
      hipFree(d_X);
  if(d_P)
      hipFree(d_P);

  if(d_F)
      hipFree(d_F);
  if(d_Ztemp)
      hipFree(d_Ztemp);
  if(d_S)
      hipFree(d_S);
  if(d_s)
      hipFree(d_s);
  if(d_K)
      hipFree(d_K);
  if(d_H)
      hipFree(d_H);
  if(d_E)
      hipFree(d_E);
  if(d_Ft)
      hipFree(d_Ft);
  if(d_Ht)
      hipFree(d_Ht);
  if(d_Si)
      hipFree(d_Si);
  if(d_Y)
      hipFree(d_Y);
  if(d_I)
      hipFree(d_I);
  if(d_Hint)
      hipFree(d_Hint);
  if(d_Sint)
      hipFree(d_Sint);
  if(d_Kint)
      hipFree(d_Kint);
  if(d_Xint)
      hipFree(d_Xint);
  if(d_Pint)
      hipFree(d_Pint);
  if(d_Pint2)
      hipFree(d_Pint2);

  printf("\nDevice Cleanup Successful\n");


  __cudaSafeCall(hipDeviceReset()) ;

  exit(0);

}




