//Includes
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "matrix_kernel.cu"
#include "matrix.cu"
#include "../include/libCuda.h"
#include "../src/invert.cpp"


using namespace onCuda::KalmanFilter;

C_kalman::C_kalman()
  {
  this->stat    = new hipblasStatus_t;
  this->handle  = new hipblasHandle_t;
  hipblasCreate  (this->handle);
  }

C_kalman::~C_kalman()
{

}
void C_kalman::allocateMemory()
  {
    //no == zeilen
    //ns == spalten

    //Allocate vectors  in device memory
    hipMalloc(&d_X, ns*1);
    hipMalloc(&d_P, ns*ns);
    hipMalloc(&d_F, ns*ns);
    hipMalloc(&d_Z, no*1);
    hipMalloc(&d_S, no*no);
    hipMalloc(&d_s, no*no);
    hipMalloc(&d_K, ns*no);
    hipMalloc(&d_H, no*ns);
    hipMalloc(&d_E, no*no);
    hipMalloc(&d_Ft, ns*ns);
    hipMalloc(&d_Ht, ns*no);
    hipMalloc(&d_Si, no*no);
    hipMalloc(&d_Y, no*1);
    hipMalloc(&d_I, ns*ns);
    hipMalloc(&d_Hint, no*ns);
    hipMalloc(&d_Sint, no*no);
    hipMalloc(&d_Kint, ns*no);
    hipMalloc(&d_Xint, ns*1);
    hipMalloc(&d_Pint, ns*ns);
    hipMalloc(&d_Pint2, ns*ns);
    lda = ((no+15)&~15|16);


    hipError_t ret = hipHostMalloc( (void**)&A, no*lda*sizeof(float) );
    if( ret != hipSuccess )
      {
      printf("Failed to allocate %d memory", no*lda*sizeof(float));
      return;
      }//if
    printf("\nAllocation of the Device memory completed...\n");


    for(int i = 0; i<no ; i++)
      {
      for(int j = i; j < no; j++)
        {
        A[i*lda + j] = s[i * no + j];
        A[j*lda+i] = A[i*lda+j];
        }//for
      }//for

  }//allocateMemory()

void C_kalman::initialize (float *F, float *H,float *E,float *s, int ns, int no)
    //X = Estimate
    //P = Uncertainty Covariance Matrix
    //F = State transition Matrix - Prediciton Matrix
    //Z = Measurement
    //H = Measurement Function - observation model which maps the true state space into the observed space
    //E = Measurement/Observation noise
    //I = Identity Matrix

  {
  this->ns = ns;
  this->no = no;
  allocateMemory();

  //Init I
  //malloc(&I,ns,ns);
  float* h_I = &I;
  hipMemcpy(d_I, h_I, ns*ns, hipMemcpyHostToDevice);
  Identity(d_I, ns);

  // Copy Input vectors from host memory to device memory
  hipMemcpy(d_F, F, ns*ns, hipMemcpyHostToDevice);
  hipMemcpy(d_H, H, no*ns, hipMemcpyHostToDevice);
  hipMemcpy(d_E, E, no*no, hipMemcpyHostToDevice);

  // Set the kernel arguments
  int threadsPerBlock = 256;
  int Nos = no*ns;
  int Ns =  ns;
  int No = no;
  int Ns2 = ns*ns;
  int No2 = no*no;
  int blocksPerGridNos = (Nos + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNs = (Ns + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNo = (No + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNs2 = (Ns2 + threadsPerBlock - 1) / threadsPerBlock;
  int blocksPerGridNo2 = (No2 + threadsPerBlock - 1) / threadsPerBlock;

  //Transpose
  MatTranspose<<<blocksPerGridNos, threadsPerBlock>>>(d_Ht,d_H,no,ns);
  printf("\n Transpose of H successful\n");

  MatTranspose<<<blocksPerGridNos, threadsPerBlock>>>(d_Ft,d_F,ns,ns);
  printf("\n Transpose of F successful\n");

  //Inverse using CUBLAS

  if( hipSetDevice( dev ) != hipSuccess )
    {
    printf( "Failed to set device %d\n", dev );
    return;
    }

  if(   hipblasCreate  (this->handle) != HIPBLAS_STATUS_SUCCESS )
    {
    printf( "failed to initialize the cublas library\n" );
    return;
    }
  printf("Cublas initialized...\n");

  invert(A, lda, no, handle);

  for(int i = 0; i<no ; i++)
    {
    for(int j = i; j < no; j++)
      {
      si[i * no + j] = A[i * lda + j];
      si[j * no + i] = si[i * no + j];
       }
     }

  hipMemcpy(d_Si, si, no*no, hipMemcpyHostToDevice);

  }

void C_kalman::correct       (float *measurement)
  {

//  Update Phase
//  kalmanGain =Pk-*HT [ H*Pk-HT+Ek]-1
//  newEstimate = xk-+kalmanGain[zk �H*xk-]
//  Pk = (I �kalmanGain*H)Pk-

  //Update
  //S = HPHt + E
  //K = PHtSi
  //Y = Z-HX
  //X = X + KY
  //P = [I - KH]P

  hipMemcpy(d_F, F, ns*ns, hipMemcpyHostToDevice);
  //step 1  to calculate Y = Z - HX
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Y, d_H, d_X, no, ns);
  MatSub<<<blocksPerGridNo, threadsPerBlock>>>(d_Y, d_Z, d_Y, no, 1);

  //step 2 to calculate  S = HPHt + E
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Hint, d_H, d_P, no, ns);
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Sint, d_Hint, d_Ht, no, ns);
  MatAdd<<<blocksPerGridNo2, threadsPerBlock>>>(d_S, d_Sint, d_E, no, no);

  //step 3 to calcualte K = PHtSi
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Kint, d_P, d_Ht, no, ns);
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_K, d_Kint, d_Si, ns, no);

  //step4 to calculate  X = X+ KY
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Xint, d_K, d_Y, ns, no);
  MatAdd<<<blocksPerGridNs, threadsPerBlock>>>(d_X, d_X, d_Xint, ns, 1);

  //step5 to calculate [I - KH]P
  MatMult<<<blocksPerGridNos, threadsPerBlock>>>(d_Pint, d_K, d_H, ns, no);
  MatSub<<<blocksPerGridNs2, threadsPerBlock>>>(d_Pint, d_I, d_Pint, ns, ns);
  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_Pint2, d_Pint, d_P, ns, ns);
  MatCopy<<<blocksPerGridNs2, threadsPerBlock>>>(d_P, d_Pint2, ns, ns);

  // Host wait for the kernel to finish
  hipDeviceSynchronize();
  }

float C_kalman::predict()
{

  // Predict Phase
  //  xk-= Fk*xk-1 + Bk*uk
  //  Pk-= FkPk-1FkT+ QK

  //Prediction Phase
  // X = FX
  // P = FPFt

  //step 1 to calculate X = F*X
  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_Xint, d_F, d_X, ns, ns);
  MatCopy<<<blocksPerGridNs, threadsPerBlock>>>(d_X, d_Xint, ns, 1);

  //step2 to calculate P = FPFt
  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_Pint, d_F, d_P, ns, ns);
  MatMult<<<blocksPerGridNs2, threadsPerBlock>>>(d_P, d_Pint, d_Ft, ns, ns);
  // Host wait for the kernel to finish
  hipDeviceSynchronize();

  // Copy result from device memory to host memory
  // h_X contains the result in host memory
  hipMemcpy(h_X, d_X, ns*1, hipMemcpyDeviceToHost);
  float estimate;
  estimate = *h_X;
  return estimate;
}


void C_kalman::cleanup()
  {
  // stop and destroy timer
  hipHostFree( A );
  hipblasDestroy(*handle);

  // Free host memory
  if(X)
      free(X);
  if(P)
      free(P);
  if(F)
      free(F);
  if(Ztemp)
      free(Ztemp);
  if(S)
      free(S);
  if(s)
      free(s);
  if(K)
      free(K);
  if(H)
      free(H);
  if(E)
      free(E);
  if(Ft)
      free(Ft);
  if(Ht)
      free(Ht);
  if(Si)
      free(Si);
  if(Y)
      free(Y);
  if(Hint)
      free(Hint);
  if(Sint)
      free(Sint);
  if(Kint)
      free(Kint);
  if(Xint)
      free(Xint);
  if(Pint)
      free(Pint);
  if(Pint2)
      free(Pint2);
  //printf("\n Host Cleanup Successful\n");

  // Free device memory

  if(d_X)
      hipFree(d_X);
  if(d_P)
      hipFree(d_P);

  if(d_F)
      hipFree(d_F);
  if(d_Ztemp)
      hipFree(d_Ztemp);
  if(d_S)
      hipFree(d_S);
  if(d_s)
      hipFree(d_s);
  if(d_K)
      hipFree(d_K);
  if(d_H)
      hipFree(d_H);
  if(d_E)
      hipFree(d_E);
  if(d_Ft)
      hipFree(d_Ft);
  if(d_Ht)
      hipFree(d_Ht);
  if(d_Si)
      hipFree(d_Si);
  if(d_Y)
      hipFree(d_Y);
  if(d_I)
      hipFree(d_I);
  if(d_Hint)
      hipFree(d_Hint);
  if(d_Sint)
      hipFree(d_Sint);
  if(d_Kint)
      hipFree(d_Kint);
  if(d_Xint)
      hipFree(d_Xint);
  if(d_Pint)
      hipFree(d_Pint);
  if(d_Pint2)
      hipFree(d_Pint2);

  printf("\nDevice Cleanup Successful\n");


  hipDeviceReset() ;

  exit(0);

}




