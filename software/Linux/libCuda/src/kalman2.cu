#include "hip/hip_runtime.h"
#include "../include/libCuda.h"


#define gravity 9.807f
#define MeasureSize 6 * sizeof(float)
#define PredictSize  3 * sizeof(float)
#define CovSize 16 * sizeof(float)


using namespace onCuda::KalmanFilter2;


/************************************* Variablendefinition und -deklaration ************************************/

/************************* DEVICE VARIABLEN ***************************/

__device__  float H[18]
{
1.0,   0,   0, 0, 0, 0,
  0, 1.0,   0, 0, 0, 0,
  0,   0, 1.0, 0, 0, 0

};

__device__  float HT[18]    =
  {
  1.0,   0,   0,
    0, 1.0,   0,
    0,   0, 1.0,
    0,   0,   0,
    0,   0,   0,
    0,   0,   0,
  };

__device__  float A[36] =
  {
    1.0,   0,    0,  999,   0,    0,
      0, 1.0,    0,    0, 999,    0,
      0,   0,  1.0,    0,   0,  999,
      0,   0,    0,  1.0,   0,    0,
      0,   0,    0,    0, 1.0,    0,
      0,   0,    0,    0,   0,  1.0
  };

__device__  float AT[36] =
  {
    1.0,   0,    0,    0,   0,    0,
      0, 1.0,    0,    0,   0,    0,
      0,   0,  1.0,    0,   0,    0,
    999,   0,    0,  1.0,   0,    0,
      0, 999,    0,    0, 1.0,    0,
      0,   0,  999,    0,   0,  1.0
  };

__device__  float B[6] =
{
0,
999,
0,
0,
999,
0
};

__device__  float u[1]      = { gravity};

__device__  float Q[16]     =
  {
     0, 0.01,     0,    0,
  0.01,    0,     0,    0,
     0,    0, 0.002, 0.01,
     0,    0,  0.01, 0.001
  };

__device__  float R[4]      =
  {
  0.01, 0.01,
  0.01, 0.01
  };

__device__  float I[16]     =
  {
  1.0, 0, 0, 0,
  0, 1.0, 0, 0,
  0, 0, 1.0, 0,
  0, 0, 0, 1.0
  };

/************************* HOST VARIABLEN ***************************/
float h_A[36]   =
  {
  1.0,   0,    0,  999,   0,    0,
  0, 1.0,    0,    0, 999,    0,
  0,   0,  1.0,    0,   0,  999,
  0,   0,    0,  1.0,   0,    0,
  0,   0,    0,    0, 1.0,    0,
  0,   0,    0,    0,   0,  1.0
  };
float h_AT[36]  =
  {
  1.0,   0,    0,    0,   0,    0,
  0, 1.0,    0,    0,   0,    0,
  0,   0,  1.0,    0,   0,    0,
  999,   0,    0,  1.0,   0,    0,
  0, 999,    0,    0, 1.0,    0,
  0,   0,  999,    0,   0,  1.0
  };
float h_B[36]   =
  {
  0,
  999,
  0,
  0,
  999,
  0
  };

/************************************* KERNEL FUNKTIONEN ************************************/

__device__ float  ele_multi       (float* A,        float* B,    int Awidth,  int Bwidth, int tx, int ty)
  {
  //calculate one element of the product of two matrices
  float Pvalue = 0;
  for (int k = 0; k < Awidth; ++k){
    float Melement = A[ty * Awidth + k];
    float Nelement = B[k * Bwidth + tx];
    Pvalue += Melement * Nelement;
  }
  return Pvalue;
  }
__global__ void   PredictKernel   (float* predictD, float* covD,      float* new_predictD, float* new_covD, int point_num)
  {
  //caculate x_k' = A * x_{k-1} + B*u
  //calculate P_k' = A * P_{k-1} * A^T + Q

  //Kernel function for the first two steps of Kalman Filter
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;

    __shared__ float temp[CovSize];
    float value1, value2;

    //caculate x_k' = A * x_{k-1} + B*u
    if (tx < 1){
      value1 = ele_multi(A, predictD + bx * PredictSize, 6, 1, tx, ty);
      value2 = ele_multi(B, u, 1, 1, tx, ty);
      new_predictD[bx * PredictSize + ty] = value1 + value2;
    }

    //calculate P_k' = A * P_{k-1} * A^T + Q
    value1 = ele_multi(A, covD + bx * CovSize, 4, 4, tx, ty);
    temp[ty * 4 + tx] = value1;

    __syncthreads();

    value1 = ele_multi(temp, AT, 4, 4, tx, ty);
    if (bx < point_num)
      new_covD[bx * CovSize + ty * 4 + tx] = value1 + Q[ty * 4 + tx];

    __syncthreads();
  }
__global__ void   UpdateKernel    (float* dataD,    float* predictD,  float* covD, float* new_predictD, float* new_covD, int point_num, int ite_num)
  {
  //kernel functino for the left three steps of Kalman Filter
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;

    float value;

    __shared__ float temp[CovSize];

    //calculate H*P
    if (ty < 2){
      value = ele_multi(H, covD + bx*CovSize, 4, 4, tx, ty);
      temp[ty * 4 + tx] = value;
    }

    __syncthreads();

    //calculate H*P_k*H^T + R
    __shared__ float temp2[PredictSize];
    if (ty < 2 && tx < 2){
      value = ele_multi(temp, HT, 4, 2, tx, ty);
      temp2[ty * 2 + tx] = value + R[ty * 2 + tx];
    }

    //calculate P_k* H^T
    __shared__ float temp3[8];
    if (tx < 2){
      value = ele_multi(covD + bx*CovSize, HT, 4, 2, tx, ty);
      temp3[ty * 2 + tx] = value;
    }

    __syncthreads();

    //calculate K
    __shared__ float K[8];
    float det = temp2[0] * temp2[3] - temp2[2] * temp2[1];
    __shared__ float temp2_inv[4];
    temp2_inv[0] = 1.0f / det * temp2[3];
    temp2_inv[1] = -1.0f / det * temp2[1];
    temp2_inv[2] = -1.0f / det * temp2[2];
    temp2_inv[3] = 1.0f / det * temp2[0];
    if (tx < 2){
      value = ele_multi(temp3, temp2_inv, 2, 2, tx, ty);
      K[ty * 2 + tx] = value;
    }

    //calculate z_k - H*x_k'
    __shared__ float temp4[8];
    if (tx < 1 && ty < 2){
      value = ele_multi(H, predictD + bx * PredictSize, 4, 1, tx, ty);
      temp4[ty] = dataD[MeasureSize * bx + ty] - value;
    }

    __syncthreads();
    //calculate x_k
    if (tx < 1){
      value = ele_multi(K, temp4, 2, 1, tx, ty);
      new_predictD[bx * PredictSize + ty] = predictD[bx * PredictSize + ty] + value;
    }

    //caculate I-K*H
    __shared__ float temp5[CovSize];
    value = ele_multi(K, H, 2, 4, tx, ty);
    temp5[ty * 4 + tx] = I[ty * 4 + tx] - value;
    __syncthreads();

    //calculate P_k
    value = ele_multi(temp5, covD + bx*CovSize, 4, 4, tx, ty);
    new_covD[bx * PredictSize + ty * 4 + tx] = value;

    __syncthreads();
  }
__global__ void   initIdentityGPU (int **devMatrix, int numR, int numC)
  {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  if(y < numR && x < numC)
    {
    if(x == y)
      devMatrix[y][x] = 1;
    else
      devMatrix[y][x] = 0;
    }
  }
/************************************* KONSTRUKTOR - DESTRUKTOR  ************************************/
C_kalman::C_kalman                (float* h_A,int rowsA, int colsA, float* h_B, int measurementsB, float*h_H, int rowsH, int colsH)
  {
  //H = observationModel
  //A = stateTransitionModel
  //Q = CovarianceProcessNoise

  //R = CovarianceObservationNoise
  //CUBLAS INIT
  this->stat    = new hipblasStatus_t;
  this->handle  = new hipblasHandle_t;
  hipblasCreate  (this->handle);
  init          (h_A, rowsA, colsA, h_B, measurementsB, h_H, rowsH, colsH);
  }
C_kalman::~C_kalman               ()
  {
  hipblasDestroy(*handle);
  delete(handle);
  delete(stat);
  }
/************************************* CPU METHODEN ************************************/
int C_kalman::init                (float* h_A,int rowsA, int colsA, float* h_B, int measurementsB, float*h_H, int rowsH, int colsH)
  {
  allocateFloat(A, rowsA, colsA);
  allocateFloat(B, measurementsB, 1);
  allocateFloat(H, rowsH, colsH);

  //SET A
  *stat = hipblasSetMatrix (rowsA, colsA, sizeof(float), &(h_A[0]), colsA, A, colsA);
  if (*stat != HIPBLAS_STATUS_SUCCESS)
    {
    printf ("Cublas set Matrix A failed");
    hipFree (A);
    hipblasDestroy(*handle);
    return EXIT_FAILURE;
    }

  //SET B
  *stat = hipblasSetVector(measurementsB,sizeof(float) ,&(h_B[0]), 1, B, 1);
  if (*stat != HIPBLAS_STATUS_SUCCESS)
    {
    printf ("Cublas set Matrix A failed");
    hipFree (B);
    hipblasDestroy(*handle);
    return EXIT_FAILURE;
    }

  //SET H
  *stat = hipblasSetMatrix (rowsH, colsH, sizeof(float), &(h_H[0]), colsH, H, colsH);
  if (*stat != HIPBLAS_STATUS_SUCCESS)
    {
    printf ("Cublas set Matrix A failed");
    hipFree (H);
    hipblasDestroy(*handle);
    return EXIT_FAILURE;
    }

    float const alpha(1.0);
    float const beta(0.0);

    //Transpose matrix AT + HT
    hipblasSgeam(*handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rowsA, colsA, &alpha, &(A[0]), colsA, &beta, &(A[0]), rowsA, &(AT[0]), rowsA );
    hipblasSgeam(*handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rowsH, colsH, &alpha, &(H[0]), colsH, &beta, &(H[0]), rowsH, &(HT[0]), rowsH );

 }
int C_kalman::allocateFloat       (float* devPtrA, int m/*column*/, int n/*row*/)
  {
  hipMalloc((void**)devPtrA, m*n*sizeof(float));
  }
void C_kalman::rowToColumnMajor   (float* Input, float* Output, int rows, int cols)
  {
  for (int row = 0; row < rows; row++)
    {
    for (int col = 0; col < cols; col++)
      {
      /* Column-major order */
      float d = Input[col*rows +row];

      if (d < 0) { d = 0; }

      /* Row-major order */
      /* Map from x=0,y=0 at buttom left corner to
         0,0 at top left corner */
      Output[cols*(rows-1 -row) + col] = d;
      }
    }
  }

int C_kalman::setQ               (float* h_Q, int rowsQ, int colsQ)
  {
  *stat = hipblasSetMatrix (rowsQ, colsQ, sizeof(float), &(h_Q[0]), colsQ, Q, colsQ);
  if (*stat != HIPBLAS_STATUS_SUCCESS)
    {
    printf ("Cublas set Matrix A failed");
    hipFree (Q);
    hipblasDestroy(*handle);
    return EXIT_FAILURE;
    }
  }
int C_kalman::setR               (float* h_R, int rowsR, int colsR)
  {
  *stat = hipblasSetMatrix (rowsR, colsR, sizeof(float), &(h_R[0]), colsR, R, colsR);
  if (*stat != HIPBLAS_STATUS_SUCCESS)
    {
    printf ("Cublas set Matrix A failed");
    hipFree (R);
    hipblasDestroy(*handle);
    return EXIT_FAILURE;
    }
  }


void C_kalman::predict_single     (float* predict, float* covD, float* new_predict, float* new_covD, int point_num, float delta_t)
  {
  //x_k' = A * x_{k-1} + B*u
  //P_k' = A * P_{k-1} * A^T + Q

  float* predictD, *new_predictD;

  for(int i=0; i < 36; i++)
    {
      if(h_A[i] == 999)
        h_A[i] = delta_t;
      if(h_AT[i] == 999)
        h_AT[i] = delta_t;

    }
  h_B[1] = 0.5f*(delta_t*delta_t);
  h_B[4] = delta_t;

  hipMemcpy(A, h_A, 36 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(AT, h_AT, 36 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B, h_B, 36 * sizeof(float), hipMemcpyHostToDevice);

  hipMalloc(&predictD, point_num* PredictSize* sizeof(float));
  hipMalloc(&new_predictD, point_num* PredictSize* sizeof(float));

  hipMemcpy(predictD, predict, point_num*PredictSize*sizeof(float), hipMemcpyHostToDevice);
  dim3 dimBlock(4, 4);
  dim3 dimGrid(point_num, 1);

  PredictKernel<<<dimGrid,dimBlock >>>(predictD, covD, new_predictD, new_covD, point_num);

  // After this step, data in PredictData is x', data in Covariance is P'
  hipMemcpy(new_predict, new_predictD, point_num*PredictSize*sizeof(float), hipMemcpyDeviceToHost);
  hipFree(predictD);
  hipFree(new_predictD);
  }
void C_kalman::update_single      (float* data, float* predict, float* covD, float* new_predict, float* new_covD, int point_num, float delta_t, int ite_num)
  {
  //the left three steps of Kalman Filter
  // H*P
  // H*P_k*H^T + R
  //calculate P_k* H^T
  //calculate K
  //calculate z_k - H*x_k'
  //calculate x_k
  //caculate I-K*H
  //calculate P_k

  float* predictD,  *new_predictD, *dataD;

  hipMalloc(&predictD, point_num* PredictSize* sizeof(float));
  hipMalloc(&new_predictD, point_num* PredictSize* sizeof(float));
  hipMalloc(&dataD, point_num * 2 * sizeof(float));

  hipMemcpy(predictD, predict, point_num*PredictSize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dataD, data, point_num * 2 * sizeof(float), hipMemcpyHostToDevice);

  dim3 dimBlock(4, 4);
  dim3 dimGrid(point_num, 1);
  UpdateKernel << <dimGrid, dimBlock >> >(dataD, predictD, covD, new_predictD, new_covD, point_num, ite_num);

  // After this step, data in PredictData is x, data in Covariance is P
  hipMemcpy(new_predict, new_predictD, point_num*PredictSize*sizeof(float), hipMemcpyDeviceToHost);

  hipFree(predictD);
  hipFree(new_predictD);
  hipFree(dataD);
  }




